#include "hip/hip_runtime.h"
#include <mpi.h>
#include <omp.h>

#include <chrono>
#include <cmath>
#include <cstdio>
#include <vector>
using namespace std;

__global__ void matmul(float* A, float* B, float* C, int N, int offset) {
  int i = blockIdx.y;
  int j = threadIdx.x + blockDim.x * blockIdx.x;
  float sum = 0.0f;
  extern __shared__ float A_s[];
  for (int ks = 0; ks < N; ks += blockDim.x) {
    __syncthreads();
    A_s[threadIdx.x] = A[N * i + ks + threadIdx.x];
    __syncthreads();
    for (int k = ks; k < ks + blockDim.x; k++) {
      sum += A_s[k - ks] * B[N * k + j];
    }
  }
  C[N * i + j + offset] = sum;
}

int main(int argc, char** argv) {
  int size, rank;
  MPI_Init(&argc, &argv);
  MPI_Comm_size(MPI_COMM_WORLD, &size);
  MPI_Comm_rank(MPI_COMM_WORLD, &rank);

  int gpusize, gpurank;
  hipGetDeviceCount(&gpusize);
  hipSetDevice(rank % gpusize);
  hipGetDevice(&gpurank);

  int N = 4096;
  int M = 1024;
  vector<float> A(N * N);
  vector<float> B(N * N);
  vector<float> C(N * N, 0);
  for (int i = 0; i < N; i++) {
    for (int j = 0; j < N; j++) {
      A[N * i + j] = drand48();
      B[N * i + j] = drand48();
    }
  }

  float* subA;
  float* subB;
  float* subC;
  float* recv;
  hipMallocManaged(&subA, N * N / size);
  hipMallocManaged(&subB, N * N / size);
  hipMallocManaged(&subC, N * N / size);
  for (int i = 0; i < N * N / size; i++) {
    subC[i] = 0;
  }

  int offset = N / size * rank;
  for (int i = 0; i < N / size; i++) {
    for (int j = 0; j < N; j++) {
      subA[N * i + j] = A[N * (i + offset) + j];
    }
  }
  for (int i = 0; i < N; i++) {
    for (int j = 0; j < N / size; j++) {
      subB[N / size * i + j] = B[N * i + j + offset];
    }
  }

  int recv_from = (rank + 1) % size;
  int send_to = (rank - 1 + size) % size;

  double comp_time = 0, comm_time = 0;
  for (int irank = 0; irank < size; irank++) {
    auto tic = chrono::steady_clock::now();
    offset = N / size * ((rank + irank) % size);

    dim3 grid(N / M, N);
    matmul<<<grid, M, M * sizeof(float)>>>(subA, subB, subC, N, offset);

    auto toc = chrono::steady_clock::now();
    comp_time += chrono::duration<double>(toc - tic).count();
    MPI_Request request[2];
    MPI_Isend(&subB[0], N * N / size, MPI_FLOAT, send_to, 0, MPI_COMM_WORLD,
              &request[0]);
    MPI_Irecv(&recv[0], N * N / size, MPI_FLOAT, recv_from, 0, MPI_COMM_WORLD,
              &request[1]);
    MPI_Waitall(2, request, MPI_STATUS_IGNORE);
#pragma omp parallel for
    for (int i = 0; i < N * N / size; i++) {
      subB[i] = recv[i];
    }
    tic = chrono::steady_clock::now();
    comm_time += chrono::duration<double>(tic - toc).count();
  }
  MPI_Allgather(&subC[0], N * N / size, MPI_FLOAT, &C[0], N * N / size,
                MPI_FLOAT, MPI_COMM_WORLD);

#pragma omp parallel for collapse(2)
  for (int i = 0; i < N; i++) {
    for (int j = 0; j < N; j++) {
      for (int k = 0; k < N; k++) {
        C[N * i + j] -= A[N * i + k] * B[N * k + j];
      }
    }
  }
  double err = 0;
  for (int i = 0; i < N; i++) {
    for (int j = 0; j < N; j++) {
      err += fabs(C[N * i + j]);
    }
  }
  if (rank == 0) {
    double time = comp_time + comm_time;
    printf("N    : %d\n", N);
    printf("comp : %lf s\n", comp_time);
    printf("comm : %lf s\n", comm_time);
    printf("total: %lf s (%lf GFlops)\n", time, 2. * N * N * N / time / 1e9);
    printf("error: %lf\n", err / N / N);
  }
  MPI_Finalize();
}